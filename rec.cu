
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <math.h>
#include <vector> 
#include <time.h> 

using namespace std; 




__global__ void tryy(float *d_engrec,float *d_xrec,float *d_yrec, float *d_xx, float *d_yy, float *d_engg, float *d_inx, int blocks){

int is,il;
int count2;
int globalIndex= (blockIdx.x * blocks) + threadIdx.x;
 is= d_inx[globalIndex];
 il= d_inx[globalIndex+1];
 count2=is;
for(int j=is;j<il;j++){
    if((d_yy[j]<(d_yy[j+1]-1))||(d_yy[j]==d_yy[j+1]&& d_xx[j]<(d_xx[j+1]-2))){
                        d_xrec[count2]=d_xx[j];
                          d_yrec[count2]=d_yy[j];
                          d_engrec[count2]=d_engg[j];
                          count2++;
                        }
                        
 //================================================double                       
else if  (((d_yy[j]== d_yy[j+1]) && (d_xx[j+1]== (d_xx[j]+1)) && ((d_xx[j+2]>d_xx[j+1]+1)))  || ((((d_yy[j+2]>d_yy[j+1]+1)) && ( ((d_yy[j+1]+1== d_yy[j+1])   &&   (d_xx[j]== (d_xx[j+1]))) ))))
{           d_engrec[count2]= d_engg[j]+ d_engg[j+1];
                if(d_xx[j]== d_xx[j+1] ){
                    if(d_engg[j]> d_engg[j+1]){
                        d_xrec[count2]=d_xx[j];
                        d_yrec[count2]=d_yy[j];
                    }
                   else{
                    d_xrec[count2]=d_xx[j+1];
                    d_yrec[count2]=d_yy[j+1];
                       }}
                if(d_yy[j]== d_yy[j+1]){
                   if(d_engg[j]> d_engg[j+1]){
                        d_yrec[count2]=d_yy[j];
                        d_xrec[count2]=d_xx[j];
                                             }
                    else{
                    d_yrec[count2]=d_yy[j+1];
                    d_xrec[count2]=d_xx[j+1];
                       }}
                         j++;
                        count2++;
                         }   
                         
                         
///========================================triple event recombination
else if ((d_yy[j]==d_yy[j+2]&& d_yy[j+3]>(d_yy[j+2]+1))||( d_yy[j]==(d_yy[j+2]+1)&& d_xx[j]<(d_xx[j+3]+1)))
         
         {
            d_engrec[count2]= d_engg[j]+ d_engg[j+1]+ d_engg[j+2];
             d_yrec[count2]=d_yy[j+2];
             d_xrec[count2]=d_xx[j+2];
              j++;
              j++;
            count2++;
         }

 ///==========================================quad event recombination




 else if(d_yy[j]== d_yy[j+1] && d_yy[j+2]== d_yy[j+3]/*&& y[o]<(y[o+4]-1)*/&&d_xx[j]==(d_xx[j+1]+1) ) {
            d_engrec[count2]= d_engg[j]+ d_engg[j+1]+ d_engg[j+2]+ d_engg[j+3];
            
             d_yrec[count2]=d_yy[j+2];
             d_xrec[count2]=d_xx[j+2];
             
//             if ((eng.at(o)>eng.at(o+1))&&(eng.at(o)>eng.at(o+2))&&(eng.at(o)>eng.at(o+3))){
//             x_rec.push_back(x.at(o));
//             y_rec.push_back(y.at(o));
//             }
//              if ((eng.at(o+1)>eng.at(o))&&(eng.at(o+1)>eng.at(o+2))&&(eng.at(o+1)>eng.at(o+3))){
//             x_rec.push_back(x.at(o+1));
//             y_rec.push_back(y.at(o+1));
//             }
//              if ((eng.at(o+2)>eng.at(o+1))&&(eng.at(o+2)>eng.at(o))&&(eng.at(o+2)>eng.at(o+3))){
//             x_rec.push_back(x.at(o+2));
//             y_rec.push_back(y.at(o+2));
//             }
//              if ((eng.at(o+3)>eng.at(o+1))&&(eng.at(o+3)>eng.at(o+2))&&(eng.at(o+3)>eng.at(o))){
//             x_rec.push_back(x.at(o+3));
//             y_rec.push_back(y.at(o+3));
//             }
             //cout << "quad"<< endl;
              //cout << x_rec.at(o)<< y_rec.at(o)<<endl;
              j++;
              j++;
              j++;
            count2++;
         }                        
                        
    
}}
    
    




int main(){


ifstream file( "c12_siegen_19072017_01", ios::in );
    string part1,part2;
    string dd= "HT";
    string dd2= "SF";
    int num1, num2, num3;
    int numb=0 ;
    int subnumb=0 ;
    int nframe=0;
    
 
   int cou=0;
    if( !file )
        cerr << "Cant open " << endl;
    while( file >> part1 )
    {
        if (part1 == dd){
        numb++;
        }
        
       if (part1 == dd2){
       nframe++;
       }
    }
       
 //===========================================================================================================================   
    float frameIndexr[nframe+1];//x[numb], y[numb] , eng[numb], 
    float *x= new float[numb]; 
    float *y= new float[numb];
    float *eng= new float[numb];
    
    frameIndexr[0]=0;
    int cou2=1;
int rf=1000;

    //cout<<"i am here  "<< numb<<endl;
  ifstream file2( "c12_siegen_19072017_01", ios::in );
   while( file2 >> part2 >>  num1 >> num2>> num3 )
    {  if (cou2>rf)break;
        if (part2 == dd){
        x[cou]= num1;
        y[cou]=num2;
        eng[cou]=num3;
       // cout<<eng[cou]<<endl;
        cou++;
	subnumb++;
   
        }
        
       if (part2 == dd2){
        frameIndexr[cou2]=frameIndexr[cou2-1]+subnumb;
        //cout<<frameIndexr[cou2]<<endl;
        subnumb=0;
        cou2++;
       }
    }    
   
    
 //===================================================================================   
    
    
    
    
    
int sizeFrame=nframe+1; 
//cout<<"  "<<sizeFrame<<"  "<< nframe<<endl;

//int x[numb],y[numb],eng[numb],frameIndex[sizeFrame];

// for (int i=0 ; i<numb ; i++){
// x[i]=xr[i];
// y[i]=yr[i];
// eng[i]=engr[i];
// }

// int count=0;
// for (int i2=1 ; i2<sizeFrame ; i2++){
// count=count+frameIndexr[i2-1];
// frameIndexr[i2]=count;
// //cout<<frameIndex[i2]<<endl;
// }

 
const int data_bytes= numb* sizeof(float); //the required memory 
const int data_bytes_2= sizeFrame * sizeof(float); 

///===========================Declaration===============================
// int h_engres[numb]; // CPU array for results
// int h_xres[numb];
// int h_yres[numb];
//cout<<"i am here  "<<endl; 

//=====================declaration of GPU
float *d_yin;
float *d_xin;
float *d_engin;
float *d_engres;
float *d_xres;
float *d_yres;
float *d_ind;
///=================== allocate GPU mem===============
hipMalloc((void **) &d_engin, data_bytes);
hipMalloc((void **) &d_engres, data_bytes);
hipMalloc((void **) &d_xres, data_bytes);
hipMalloc((void **) &d_yres, data_bytes);
hipMalloc((void **) &d_xin, data_bytes);
hipMalloc((void **) &d_yin, data_bytes);
hipMalloc((void **) &d_ind, data_bytes_2);

///================== define number of blocks with constant 1024 threads per block===========
int nthreadsperblock=32; //number of threads per block
int nblock; //number of blocks
if(sizeFrame%nthreadsperblock == 0){
nblock=sizeFrame/nthreadsperblock;
}  
else{nblock=(sizeFrame/nthreadsperblock)+1;}
//cout<< nblock << "  "<< nthreadsperblock<<endl;

///===================== copy the data to the GPU=============
hipMemcpy(d_xin, x, data_bytes, hipMemcpyHostToDevice);
hipMemcpy(d_yin, y, data_bytes, hipMemcpyHostToDevice);
hipMemcpy(d_engin, eng, data_bytes, hipMemcpyHostToDevice);
hipMemcpy(d_ind,frameIndexr, data_bytes_2, hipMemcpyHostToDevice);
clock_t   tG0=clock();
tryy<<<nblock,nthreadsperblock>>>(d_engres,d_xres,d_yres,d_xin,d_yin,d_engin,d_ind,nthreadsperblock);  

hipMemcpy(eng,d_engres, data_bytes, hipMemcpyDeviceToHost);
hipMemcpy(x,d_xres, data_bytes, hipMemcpyDeviceToHost);
hipMemcpy(y,d_yres, data_bytes, hipMemcpyDeviceToHost);
clock_t   tGf=clock();
int single=0;
for (int i2=0 ; i2<numb ; i2++){
    if(eng[i2]>0){
//cout<<eng[i2]<<"  "<<x[i2]<<"  "<<y[i2]<<endl;
        single++;
    }}

///=====================================================CPU=================================================================================================
//==========================================================================================================================================================
int frame[384][384]={{}};
   int bg[384][384]={{}};
   vector<int> xc;
   vector<int> yc;
   vector<int> engc;
vector<int> x_rec;
    vector<int> y_rec;
    vector<int> eng_rec;
   clock_t   t1=clock();
  
numb=0;
nframe=0;
int thres =50;
ifstream file3( "c12_siegen_19072017_01", ios::in );
  

     if( !file3 ){
         cerr << "Cant open " << endl;
        }
while( file3 >> part1 >>  num1 >> num2>> num3 )
     {if (nframe>rf)break;
         if (part1 == dd){
         
         xc.push_back( num1);
         yc.push_back( num2);
         engc.push_back( num3);
         numb++;}
        if (part1 == dd2){

         nframe++;
    for (int k2=0;k2<384;k2++){
        for(int j2=0;j2<384;j2++){
    frame[j2][k2]=0;
        }}
///================================starting recombination ======================================================================



for (int i=0;i<xc.size();i++)///filling the frame matrix
 {
frame[xc[i]][yc[i]]=engc[i];
bg[xc[i]][yc[i]]=50;
}

for (int kk=1;kk<384;kk++){
for(int jj=1;jj<384;jj++){
int k= jj, j=kk; 
if (frame[j][k]>bg[j][k]){   
    
    ///================================single=======================
 if(frame[j+1][k]<bg[j+1][k] && frame[j][k+1]<bg[j][k+1] &&frame[j-1][k]<bg[j-1][k]&&frame[j][k-1]<bg[j-1][k] ){

x_rec.push_back(j);
y_rec.push_back(k);
eng_rec.push_back(frame[j][k]);
frame[j][k]=0;
}
///================================double=======================
  /////==========horizontal double============================================
 else if(frame[j+1][k]>bg[j+1][k] &&frame[j+2][k]<bg[j+2][k]&&frame[j][k+1]<bg[j][k+1] &&frame[j-1][k]<bg[j-1][k]&&frame[j][k-1]<bg[j][k-1]&&frame[j+1][k+1]<bg[j+1][k+1]&&frame[j+1][k-1]<bg[j+1][k-1] ) {
   
    eng_rec.push_back((frame[j][k]+frame[j+1][k]));
    if(frame[j][k]>frame[j+1][k]){
        x_rec.push_back(j);
        y_rec.push_back(k);
    }
    else{
        x_rec.push_back(j+1);
        y_rec.push_back(k);
    }
    frame[j][k]=0;
    frame[j+1][k]=0;}
    ////===============================vertical double ========================================
 else if(frame[j][k+1]>bg[j][k+1]&&frame[j+1][k]<bg[j+1][k] &&frame[j][k+2]<bg[j][k+2] && frame[j+1][k+1]<bg[j+1][k+1]&&frame[j-1][k]<bg[j-1][k]&&frame[j-1][k+1]<bg[j-1][k+1]&&frame[j][k-1]<bg[j][k-1]) {
    
     eng_rec.push_back((frame[j][k]+frame[j][k+1]));
    if(frame[j][k]>frame[j][k+1]){
        x_rec.push_back(j);
        y_rec.push_back(k);
    }
    else{
        x_rec.push_back(j);
        y_rec.push_back(k+1);
    }
    frame[j][k]=0;
    frame[j][k+1]=0;}

///================================quadrad=======================

else if(frame[j+1][k]>bg[j+1][k]&&frame[j+1][k+1]>bg[j+1][k+1]&&frame[j][k+1]>bg[j][k+1]&&frame[j+2][k]<bg[j+2][k]&&frame[j-1][k]<bg[j-1][k]&&frame[j][k-1]<bg[j][k-1]&&frame[j+1][k-1]<bg[j+1][k-1]
  && frame[j+2][k+1]<bg[j+2][k+1] && frame[j-1][k+1]<bg[j-1][k+1] && frame[j][k+2]<bg[j][k+2] && frame[j+1][k+2]<bg[j+1][k+2] )
{
   
  eng_rec.push_back((frame[j][k]+frame[j][k+1]+frame[j+1][k]+frame[j+1][k+1]));
if(frame[j][k]>frame[j+1][k]&&frame[j][k]>frame[j][k+1]&&frame[j][k]>frame[j+1][k+1]){
     x_rec.push_back(j);
    y_rec.push_back(k);
}
else if(frame[j+1][k]>frame[j][k]&&frame[j+1][k]>frame[j][k+1]&&frame[j+1][k]>frame[j+1][k+1]){
    x_rec.push_back(j+1);
    y_rec.push_back(k);}
else if(frame[j][k+1]>frame[j][k]&&frame[j][k+1]>frame[j+1][k]&&frame[j][k+1]>frame[j+1][k+1]){
    x_rec.push_back(j);
    y_rec.push_back(k+1);
}
else{

     x_rec.push_back(j+1);
     y_rec.push_back(k+1);
}
//cout<<  frame[j][k]<<"  "<<frame[j][k+1]<<"  "<<frame[j+1][k]<<"  "<<frame[j+1][k+1]<<endl;
   frame[j][k]=0;
    frame[j][k+1]=0;
   frame[j+1][k]=0;
    frame[j+1][k+1]=0;
}



//==================================================================

///================================triple L=======================

else if(frame[j+1][k+1]>thres && frame[j][k+1]>thres &&frame[j+1][k]<thres&&frame[j][k+2]<thres&&frame[j+1][k+2]<thres&&frame[j][k-1]<thres&&frame[j-1][k]<thres&&frame[j-1][k+1]<thres&&frame[j+2][k+1]<thres&&frame[j][k+1]>frame[j][k]&&frame[j][k+1]>frame[j+1][k+1])
{
   
   eng_rec.push_back((frame[j][k]+frame[j][k+1]+frame[j+1][k+1]));
    x_rec.push_back(j);
    y_rec.push_back(k+1);
    frame[j][k]=0;
    frame[j][k+1]=0;
  frame[j+1][k+1]=0;
}


///============================triple J========================================================
else if (frame[j-1][k+1]>thres && frame[j][k+1]>thres&&frame[j+1][k]<thres &&frame[j-1][k]<thres&&frame[j][k-1]<thres&&frame[j-2][k+1]<thres&&frame[j-1][k+2]<thres
&&frame[j][k+2]<thres&&frame[j+1][k+1]<thres&&frame[j][k+1]>frame[j][k]&&frame[j][k+1]>frame[j-1][k+1] )
{
  
   eng_rec.push_back((frame[j][k]+frame[j-1][k+1]+frame[j][k+1]));
    x_rec.push_back(j);
    y_rec.push_back(k+1);
    
    frame[j][k]=0;
    frame[j-1][k+1]=0;
    frame[j][k+1]=0;
}
///================================== triple F ===================================

 else if(frame[j][k+1]>thres &&frame[j+1][k]>thres&&frame[j+2][k]<thres &&frame[j][k+2]<thres&&frame[j+1][k+1]<thres&&frame[j][k-1]<thres&&
frame[j+1][k-1]<thres&&frame[j-1][k]<thres&&frame[j-1][k+1]<thres&&frame[j][k]>frame[j+1][k]&&frame[j][k]>frame[j][k+1])
{
   
   eng_rec.push_back((frame[j][k]+frame[j+1][k]+frame[j][k+1]));
    x_rec.push_back(j);
    y_rec.push_back(k);
   frame[j][k]=0;
    frame[j][k+1]=0;
   frame[j+1][k]=0;
}

///====================================== triple 7 ====================================================

 else if(frame[j+1][k]>thres &&frame[j+1][k+1]>thres&&frame[j-1][k]<thres&&frame[j][k-1]<thres&&frame[j][k+1]<thres&&frame[j+1][k+2]<thres&&frame[j+1][k-1]<thres
&&frame[j+2][k]<thres &&frame[j+2][k+1]<thres&&frame[j+1][k]>frame[j][k]&&frame[j+1][k]>frame[j+1][k+1] 
)
{
   
   eng_rec.push_back((frame[j][k]+frame[j+1][k+1]+frame[j+1][k]));
    x_rec.push_back(j+1);
    y_rec.push_back(k);
    frame[j][k]=0;
    frame[j+1][k]=0;
    frame[j+1][k+1]=0;
}

}}}
xc.clear();
yc.clear();
engc.clear();
        
    
}}
 clock_t t=clock();  









cout<<"The total number of frames= "<<nframe<<endl; 
cout<<"The total number of frames= "<<cou2<<endl;
float gpu_time =((float)(tGf-tG0))/(CLOCKS_PER_SEC);
printf ("The GPU   (%f sec).\n",gpu_time);
float cpu_time =((float)(t-t1))/(CLOCKS_PER_SEC);
printf ("The CPU   (%f sec).\n",cpu_time);
float speed_up = (cpu_time/gpu_time)/75;
printf ("SU   (%f ).\n", ceil(speed_up));
hipFree(d_yin);
hipFree(d_xin);
hipFree(d_engin);
hipFree(d_engres);
hipFree(d_xres);
hipFree(d_yres);
hipFree(d_ind);
delete[] x;
delete[] y;
delete[] eng;
return 0 ;
}
